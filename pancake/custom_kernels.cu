#include "hip/hip_runtime.h"
#include <stdio.h>
#include "custom_kernels.cuh"
#include "Pancake.h"
#include "hash_array.h"

#include <cooperative_groups/memcpy_async.h>
#include <cooperative_groups/reduce.h>
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>
#include <cuda/barrier>
namespace cg = cooperative_groups;


#define int_div_ceil(x,y) ((x + y - 1) / y)

constexpr uint32_t npow2(uint32_t v)
{
  //return v == 1 ? 1 : 1 << (64 - __lzcnt(v - 1));
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}
static_assert(npow2(3) == 4);



//https://www.apriorit.com/dev-blog/614-cpp-cuda-accelerate-algorithm-cpu-gpu
template <typename T>
__device__ void atomicMinFloat(T* const address, const T value)
{
  if(*address <= value)
  {
    return;
  }

  int* const addressAsI = (int*)address;
  int old = *addressAsI, assumed;

  do
  {
    assumed = old;
    if(__int_as_float(assumed) <= value)
    {
      break;
    }

    old = atomicCAS(addressAsI, assumed, __float_as_int(value));
  } while(assumed != old);
}

__device__ char atomicMinChar(char* address, char val)
{
  unsigned int* base_address = (unsigned int*)((size_t)address & ~3);
  unsigned int selectors[] = {0x3214, 0x3240, 0x3410, 0x4210};
  unsigned int sel = selectors[(size_t)address & 3];
  unsigned int old, assumed, min_, new_;

  old = *base_address;
  do {
    assumed = old;
    min_ = min(val, (char)__byte_perm(old, 0, ((size_t)address & 3)));
    new_ = __byte_perm(old, min_, sel);
    old = atomicCAS(base_address, assumed, new_);
  } while(assumed != old);

  return old;
}

template <typename T>
__global__ void reduceMin(int num_batch, int num_frontier, const T* __restrict__ mult_results, T* __restrict__ batch_answers)
{
  __shared__ char sharedMin;

  for(int batch_idx = blockIdx.x; batch_idx < num_batch; batch_idx += gridDim.x) {
    const T* start_results = mult_results + batch_idx * num_frontier;

    __syncthreads();

    if(0 == threadIdx.x)
    {
      sharedMin = INT8_MAX;
    }

    __syncthreads();

    T localMin = INT8_MAX;

    for(int i = threadIdx.x; i < num_frontier; i += blockDim.x)
    {
      localMin = MIN(localMin, start_results[i]);
    }
    atomicMinChar(&sharedMin, localMin);

    __syncthreads();

    if(0 == threadIdx.x)
    {
      batch_answers[batch_idx] = sharedMin;
    }
  }
}

template<typename T>
__global__
void cuda_min_kernel(int num_batch, int num_frontier, const T* __restrict__ mult_results, T* __restrict__ batch_answers)
{
  for(int batch_idx = blockIdx.x * blockDim.x + threadIdx.x, stride = blockDim.x * gridDim.x; batch_idx < num_batch; batch_idx += stride)
  {
    const T* __restrict__ start_results = mult_results + batch_idx * num_frontier;
    T min = start_results[0];
    for(int frontier_idx = 1; frontier_idx < num_frontier; ++frontier_idx)
    {
      if(start_results[frontier_idx] < min) min = start_results[frontier_idx];
    }
    batch_answers[batch_idx] = min;
  }
}

#define TILE 16
__global__
void tiled_cuda_bitwise_set_intersection(const uint32_t rows_a,//x-axis
                                         const uint32_t rows_b,//y-axis
                                         const unsigned max_a,
                                         const uint32_t* __restrict__ hash_a,
                                         const uint8_t* __restrict__ g_vals,
                                         const uint32_t* __restrict__ hash_b,
                                         uint8_t* __restrict__ results)
{
  assert(threadIdx.x < TILE);
  assert(threadIdx.y < TILE);
  assert(blockIdx.x * blockDim.x < rows_a);
  assert(blockIdx.y * blockDim.y < rows_b);

  __shared__ uint32_t sA[NUM_INTS_PER_PANCAKE][TILE];
  __shared__ uint32_t sB[TILE][NUM_INTS_PER_PANCAKE];
  uint32_t localB[NUM_INTS_PER_PANCAKE];
  uint32_t localA[NUM_INTS_PER_PANCAKE];
  volatile __shared__ uint8_t sMin[TILE][TILE];
  volatile uint8_t minVal = UINT8_MAX;

  cg::thread_block block = cg::this_thread_block();

  //for(uint32_t by = blockIdx.y; by < int_div_ceil(rows_b, NUM_INTS_PER_PANCAKE); by += gridDim.y) {
  const uint32_t output_row = blockIdx.y * blockDim.y + threadIdx.y;

  if(output_row < rows_b) {
    for(uint32_t tidx = threadIdx.x; tidx < NUM_INTS_PER_PANCAKE; tidx += blockDim.x) {
      sB[threadIdx.y][tidx] = hash_b[output_row * NUM_INTS_PER_PANCAKE + tidx];
    }
  }

  block.sync();

  if(output_row < rows_b) {
    #pragma unroll
    for(uint32_t i = 0; i < NUM_INTS_PER_PANCAKE; ++i) {
      localB[i] = sB[threadIdx.y][i];
    }
  }

  //x goes 0 to rows_a
  for(uint32_t bx = blockIdx.x; bx < max_a; bx += gridDim.x) {
    uint32_t output_col = bx * blockDim.x + threadIdx.x;
    if(output_col < rows_a) {
      for(int tidy = threadIdx.y; tidy < NUM_INTS_PER_PANCAKE; tidy += blockDim.y) {
        sA[tidy][threadIdx.x] = hash_a[output_col * NUM_INTS_PER_PANCAKE + tidy];
      }
    }
    block.sync();
    if(output_row < rows_b && output_col < rows_a) {
      #pragma unroll
      for(uint32_t i = 0; i < NUM_INTS_PER_PANCAKE; ++i) {
        localA[i] = sA[i][threadIdx.x];
      }
      constexpr Mask gap_mask;
      uint32_t tmpF = 0;
      uint32_t tmpB = 0;
      uint32_t tmpMin;
      #pragma unroll
      for(uint32_t i = 0; i < NUM_GAP_INTS; ++i) {
        uint32_t A = localA[i];
        uint32_t B = localB[i];
        tmpF += __popc(B & (A | gap_mask[i]));
        tmpB += __popc(A & (B | gap_mask[i]));
      }
      tmpMin = MIN(tmpF, tmpB);
      #pragma unroll
      for(uint32_t i = NUM_GAP_INTS; i < NUM_INTS_PER_PANCAKE; ++i) {
        uint32_t A = localA[i];
        uint32_t B = localB[i];
        tmpMin += __popc(A & B);
      }
      results[output_row * rows_a + output_col] = static_cast<uint8_t>(NUM_PANCAKES + g_vals[output_col] - tmpMin);
      //minVal = MIN(minVal, NUM_PANCAKES + g_vals[col] - tmpMin);
      //assert(minVal > GAPX);
    }
    block.sync();
  }
  
  //assert(minVal > GAPX);
  //sMin[threadIdx.y][threadIdx.x] = minVal;
  //block.sync();
  //if(threadIdx.x == 0 && output_row < rows_b) {
  //  for(uint32_t a = 1; a < TILE && blockIdx.x * blockDim.x + a < rows_a; a++) {
  //    minVal = MIN(minVal, sMin[threadIdx.y][a]);
  //    assert(minVal > GAPX);
  //  }
  //  results[output_row * gridDim.x + blockIdx.x] = minVal;
  //}
//}
}

__global__
void naive_cuda_bitwise_set_intersection(int rows_a, int rows_b, const uint32_t* __restrict__ hash_a, const uint32_t* __restrict__ g_vals,
                                         const uint32_t* __restrict__ hash_b, uint32_t* __restrict__ results)
{
  constexpr Mask gap_mask;
  for(int batch_idx = blockIdx.x * blockDim.x + threadIdx.x, max = rows_a * rows_b, stride = blockDim.x * gridDim.x; batch_idx < max; batch_idx += stride)
  {
    int col = batch_idx / rows_a;
    int row = batch_idx % rows_a;
    int tmpF = 0;
    int tmpB = 0;
    for(int i = 0; i < NUM_INTS_PER_PANCAKE; ++i) {
      uint32_t A = hash_a[row * NUM_INTS_PER_PANCAKE + i];
      uint32_t B = hash_b[col * NUM_INTS_PER_PANCAKE + i];
      tmpF += __popc(B & (A | gap_mask[i]));
      tmpB += __popc(A & (B | gap_mask[i]));
    }
    results[batch_idx] = NUM_PANCAKES + g_vals[row] - MIN(tmpF, tmpB);
  }
}

__global__ void sharedReduceMin(const uint32_t xDim, 
                                const uint32_t yDim, 
                                const uint8_t* __restrict__ mult_results, 
                                uint8_t* __restrict__ batch_answers)
{
  cg::thread_block block = cg::this_thread_block();
  volatile uint8_t minVal = UINT8_MAX;
  volatile __shared__ uint8_t sharedMin[TILE][TILE];
  assert(threadIdx.y < TILE);
  const uint32_t output_row = blockIdx.y * blockDim.y + threadIdx.y;
  if(output_row < yDim) {
    const uint8_t* __restrict__ start_row = mult_results + output_row * xDim;
    for(unsigned input_column = blockIdx.x * blockDim.x + threadIdx.x; input_column < xDim; input_column += gridDim.x) {
      minVal = MIN(start_row[input_column], minVal);
      assert(minVal > GAPX);
    }
  }
  sharedMin[threadIdx.y][threadIdx.x] = minVal;
  block.sync();
  if(threadIdx.x == 0 && output_row < yDim) {
    #pragma unroll
    for(int i = 1; i < TILE && blockIdx.x * blockDim.x + i < xDim; ++i) {
      minVal = MIN(minVal, sharedMin[threadIdx.y][i]);
      assert(minVal > GAPX);
    }
    batch_answers[output_row] = minVal;
  }
}

void bitwise_set_intersection(hipStream_t stream,
                              int rows_a,
                              int rows_b,
                              const uint32_t* __restrict__ hash_a,
                              const uint8_t* __restrict__ g_vals,
                              const uint32_t* __restrict__ hash_b,
                              uint8_t* __restrict__ mult_results,
                              uint8_t* __restrict__ d_answers)
{
  
  //constexpr int threadsPerBlock = 256;
  //int blocksPerGrid = (rows_a * rows_b + threadsPerBlock - 1) / threadsPerBlock;
  //naive_cuda_bitwise_set_intersection << <blocksPerGrid, threadsPerBlock, 0, stream >> > (rows_a, rows_b, hash_a, g_vals, hash_b, mult_results);
  constexpr uint32_t MAX_BLOCKS_X = 1024;
  constexpr uint32_t MAX_BLOCKS_Y = 65535;
  constexpr uint32_t THREADS_X = TILE;
  constexpr uint32_t THREADS_Y = TILE;
  dim3 threadsPerBlock(MIN(rows_a, THREADS_X), MIN(rows_b, THREADS_Y), 1);
  uint32_t gridDimX = MIN(MAX_BLOCKS_X, int_div_ceil(rows_a, threadsPerBlock.x));
  uint32_t gridDimY = MIN(MAX_BLOCKS_Y, int_div_ceil(rows_b, threadsPerBlock.y));
  int max_a = int_div_ceil(rows_a, threadsPerBlock.x);
  assert(gridDimY < 65535);
  dim3 blocksPerGrid(gridDimX, gridDimY, 1);
  
  //TODO: DO THIS WHERE IT SHOULD BE DONE
  //hipMalloc(mult_results, rows_b * gridDimY * sizeof(uint8_t));

  tiled_cuda_bitwise_set_intersection << <blocksPerGrid, threadsPerBlock, 0, stream >> > (rows_a, rows_b, max_a, hash_a, g_vals, hash_b, mult_results);

  threadsPerBlock = dim3(TILE, TILE, 1);
  blocksPerGrid = dim3(int_div_ceil(gridDimX, threadsPerBlock.x), gridDimY, 1);
  //sharedReduceMin <<<blocksPerGrid, threadsPerBlock, 0, stream >>> (gridDimX, rows_b, mult_results, d_answers);
  reduceMin << <96, 16384, 0, stream >> > (rows_b, rows_a, mult_results, d_answers);
}

void reduce_min(hipStream_t stream, int num_batch, int num_frontier, const uint8_t* __restrict__ mult_results, uint8_t* __restrict__ d_batch_answers)
{
  constexpr int threadsPerBlock = 96;
  int blocksPerGrid = MIN(int_div_ceil(num_batch * num_frontier, threadsPerBlock), 16384);
  reduceMin << <blocksPerGrid, threadsPerBlock, 0, stream >> > (num_batch, num_frontier, mult_results, d_batch_answers);
}

